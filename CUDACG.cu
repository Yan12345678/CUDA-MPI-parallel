#include "hip/hip_runtime.h"
//compile with nvcc CUDACG.cu -lcublas -lcusparse
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipsparse.h"
#include "hipblas.h"
#include <time.h>
#include <stdlib.h>
#include <sys/time.h>

#define CLEANUP(s){\
		printf("%s\n",s);\
		if(h_valA) free(h_valA);\
		if(h_csrRowPtrA) free(h_csrRowPtrA);\
		if(h_csrColIndA) free(h_csrColIndA);\
		if(valA) hipFree(valA);\
		if(csrRowPtrA) hipFree(csrRowPtrA);\
		if(csrColIndA) hipFree(csrColIndA);\
		if(h_b) free(h_b);\
		if(b) hipFree(b);\
		if(r) hipFree(r);\
		if(p) hipFree(p);\
		if(Ap) hipFree(Ap);\
		if(h_x) free(h_x);\
		if(x) hipFree(x);\
		if(dBuffer) hipFree(dBuffer);\
		if(handle1) hipblasDestroy(handle1);\
		if(descrA) hipsparseDestroyMatDescr(descrA);\
		if(handle2) hipsparseDestroy(handle2);\
		if(matA) hipsparseDestroySpMat(matA);\
    	if(vecp) hipsparseDestroyDnVec(vecp);\
    	if(vecAp) hipsparseDestroyDnVec(vecAp);\
		hipDeviceReset();\
}

double cpuSecond(){
	struct timeval tp;//struct timeval in Linux
	gettimeofday(&tp,NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
}

int main(){
	//initialization
	hipError_t cudaStat1, cudaStat2, cudaStat3;

	double *h_valA = 0;
	int *h_csrRowPtrA = 0;
	int *h_csrColIndA = 0;
	double *valA = 0;
	int *csrRowPtrA = 0;
	int *csrColIndA = 0;
	double *h_b = 0;
	double *b = 0;
	double *r = 0;
	double *p = 0;
	double *Ap = 0;
	double *h_x = 0;
	double *x = 0;
	void *dBuffer = NULL;
	size_t bufferSize = 0;

	hipblasStatus_t status1;
	hipblasHandle_t handle1 = 0;
	hipsparseStatus_t status2;
	hipsparseHandle_t handle2 = 0;
	hipsparseMatDescr_t descrA = 0;
	hipsparseDnVecDescr_t vecp = 0;
	hipsparseDnVecDescr_t vecAp = 0;
	hipsparseSpMatDescr_t matA = 0;

	int n,nnz;
	double dzero = 0.0;
	double done = 1.0;

	/* use CG to slove following linear system:
	Ax = b;
	A =|3 0 2|   b=|3.5|    
	   |0 2 0|     |1.5|
	   |2 0 1|     |2.0|
	correct result x:
	x =|0.50|
	   |0.75|
	   |1.00|*/
	n = 3;
	nnz = 5;

	//which device(GPU) to use
	cudaStat1 = hipSetDevice(0);
	if(cudaStat1 != hipSuccess){
		CLEANUP("Device Set failed");
		return EXIT_FAILURE;//define in stdlib.h, has a value of 1
	}

	//matrix A(csr format) in host
	h_valA = (double*)malloc(nnz*sizeof(double));
	h_csrRowPtrA = (int*)malloc((n+1)*sizeof(int));
	h_csrColIndA = (int*)malloc(nnz*sizeof(int));
	if(!h_valA||!h_csrColIndA||!h_csrRowPtrA){
		CLEANUP("Host malloc failed(A)");
		return EXIT_FAILURE;
	}

	h_valA[0] = 3.0;
	h_valA[1] = 2.0;
	h_valA[2] = 2.0;
	h_valA[3] = 2.0;
	h_valA[4] = 1.0;

	h_csrRowPtrA[0] = 0;
	h_csrRowPtrA[1] = 2;
	h_csrRowPtrA[2] = 3;
	h_csrRowPtrA[3] = 5;

	h_csrColIndA[0] = 0;
	h_csrColIndA[1] = 2;
	h_csrColIndA[2] = 1;
	h_csrColIndA[3] = 0;
	h_csrColIndA[4] = 2;

	//matrix A(csr format) in device
	cudaStat1 = hipMalloc((void**)&valA, nnz*sizeof(double));
	cudaStat2 = hipMalloc((void**)&csrRowPtrA, (n+1)*sizeof(int));
	cudaStat3 = hipMalloc((void**)&csrColIndA, nnz*sizeof(int));
	if(cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess){
		CLEANUP("device malloc failed(A)");
		return EXIT_FAILURE;
	}

	cudaStat1 = hipMemcpy(valA, h_valA, (size_t)(nnz*sizeof(double)),hipMemcpyHostToDevice);
	cudaStat2 = hipMemcpy(csrRowPtrA, h_csrRowPtrA, (size_t)((n+1)*sizeof(int)),hipMemcpyHostToDevice);
	cudaStat3 = hipMemcpy(csrColIndA, h_csrColIndA, (size_t)(nnz*sizeof(int)),hipMemcpyHostToDevice);
	if(cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess){
		CLEANUP("memcpy from host to device failed(A)");
		return EXIT_FAILURE;
	}

	//vector b in host
	h_b = (double*)malloc(n*sizeof(double));
	h_b[0] = 3.5;
	h_b[1] = 1.5;
	h_b[2] = 2.0;

	//vector b in device
	cudaStat1 = hipMalloc((void**)&b,n*sizeof(double));
	if(cudaStat1 != hipSuccess){
		CLEANUP("device malloc failed(b)");
		return EXIT_FAILURE;
	}

	cudaStat1 = hipMemcpy(b, h_b, (size_t)(n*sizeof(double)), hipMemcpyHostToDevice);
	if(cudaStat1 != hipSuccess){
		CLEANUP("memcpy from host to device failed(b)");
		return EXIT_FAILURE;
	}

	//vector r in device
	cudaStat1 = hipMalloc((void**)&r,n*sizeof(double));
	if(cudaStat1 != hipSuccess){
		CLEANUP("device malloc failed(r)");
		return EXIT_FAILURE;
	}

	//vector p in device
	cudaStat1 = hipMalloc((void**)&p,n*sizeof(double));
	if(cudaStat1 != hipSuccess){
		CLEANUP("device malloc failed(p)");
		return EXIT_FAILURE;
	}

	//vector Ap in device
	cudaStat1 = hipMalloc((void**)&Ap,n*sizeof(double));
	if(cudaStat1 != hipSuccess){
		CLEANUP("device malloc failed(Ap)");
		return EXIT_FAILURE;
	}

	//vector x in host
	h_x = (double*)malloc(n*sizeof(double));
	for(int i=0; i<n; ++i)
		h_x[i] = 0.0;

	//vector x in device
	cudaStat1 = hipMalloc((void**)&x, n*sizeof(double));
	if(cudaStat1 != hipSuccess){
		CLEANUP("device malloc failed(x)");
		return EXIT_FAILURE;
	}	

	//initialize cublas library
	status1 = hipblasCreate(&handle1);
	if(status1 != HIPBLAS_STATUS_SUCCESS){
		CLEANUP("CUBLAS library initialize failed");
		return EXIT_FAILURE;
	}

	//initialize cusparse library
	status2 = hipsparseCreate(&handle2);
	if(status2 != HIPSPARSE_STATUS_SUCCESS){
		CLEANUP("CUSPARSE library initialize failed");
		return EXIT_FAILURE;
	}

	//initialize matrix descriptor
	status2 = hipsparseCreateMatDescr(&descrA);
	if(status2 != HIPSPARSE_STATUS_SUCCESS){
		CLEANUP("Matrix descriptor initialize failed");
		return EXIT_FAILURE;
	}
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
	
	//initialize something for hipsparseSpMV	
    // Create sparse matrix A in CSR format
    status2 = hipsparseCreateCsr(&matA, n, n, nnz,
                    csrRowPtrA, csrColIndA, valA,
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
	if(status2 != HIPSPARSE_STATUS_SUCCESS){
		CLEANUP("Create csr failed(matA)");
		return EXIT_FAILURE;
	}	    

	// Create dense vector p
	status2 = hipsparseCreateDnVec(&vecp, n, p, HIP_R_64F);
	if(status2 != HIPSPARSE_STATUS_SUCCESS){
		CLEANUP("Create DnVec failed(p)");
		return EXIT_FAILURE;
	}	
	// Create dense vector Ap
	status2 = hipsparseCreateDnVec(&vecAp, n, Ap, HIP_R_64F);
	if(status2 != HIPSPARSE_STATUS_SUCCESS){
		CLEANUP("Create DnVec failed(Ap)");
		return EXIT_FAILURE;
	}	

	//CG
	//1:
	//max iteration number = 2000
	//relative residual = 1e-7
	//compute initial residual r = b - Ax_0
	//x_0 =|0|
	//     |0|
	//     |0|
	//initial p = r
	int maxit = 2000;
	double tol = 1e-7;
	double alpha, beta, rhop, rho;
	//r = b, because x = [0;0;0] so Ax=x
	status1 = hipblasDcopy(handle1,n,b,1,r,1);
	if(status1 != HIPBLAS_STATUS_SUCCESS){
		CLEANUP("vector copy failed(r)");
		return EXIT_FAILURE;
	}	

	//p = b, because x = [0;0;0] so Ax=x
	status1 = hipblasDcopy(handle1,n,b,1,p,1);
	if(status1 != HIPBLAS_STATUS_SUCCESS){
		CLEANUP("vector copy failed(p)");
		return EXIT_FAILURE;
	}	

	status1 = hipblasDnrm2(handle1,n,r,1,&rho);//L2 fanshu
	if(status1 != HIPBLAS_STATUS_SUCCESS){
		CLEANUP("compute norm2 failed(r)");
		return EXIT_FAILURE;
	}		
	rho = rho*rho;

	//2:repeat until convergence
	for(int i=0; i<maxit; ++i)
	{

		// allocate an external buffer if needed
		status2 = hipsparseSpMV_bufferSize(handle2,
			HIPSPARSE_OPERATION_NON_TRANSPOSE,&done,
			matA,vecp,&dzero,vecAp,HIP_R_64F,HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);
		if(status2 != HIPSPARSE_STATUS_SUCCESS){
			CLEANUP("compute bufferSize failed(SpMV)");
			return EXIT_FAILURE;
		}

		cudaStat1 = hipMalloc(&dBuffer, bufferSize);
		if(cudaStat1 != hipSuccess){
			CLEANUP("device malloc failed(dBuffer)");
			return EXIT_FAILURE;
		}		

		//execute SpMV
		status2 = hipsparseSpMV(handle2, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE,&done,
			matA,vecp,&dzero,vecAp,
			HIP_R_64F,HIPSPARSE_MV_ALG_DEFAULT,dBuffer);
		if(status2 != HIPSPARSE_STATUS_SUCCESS){
			CLEANUP("compute mv failed(Ap)");
			return EXIT_FAILURE;
		}		

		status2 = hipsparseDnVecGetValues(vecAp, (void**)(&Ap));
		if(status2 != HIPSPARSE_STATUS_SUCCESS){
			CLEANUP("output vecAp failed");
			return EXIT_FAILURE;
		}	

		double tmp;
		status1 = hipblasDdot(handle1,n,p,1,Ap,1,&tmp);
		if(status1 != HIPBLAS_STATUS_SUCCESS){
			CLEANUP("compute dot failed(tmp)");
			return EXIT_FAILURE;
		}	

		//alpha = (r^T*r)/(p^T*A*p)
		alpha = rho/tmp;

		//x = x + alpha*p
		status1 = hipblasDaxpy(handle1,n,&alpha,p,1,x,1);
		if(status1 != HIPBLAS_STATUS_SUCCESS){
			CLEANUP("compute axpy failed(tmp)");
			return EXIT_FAILURE;
		}	
  		//r = r - alpha*A*p
		double tmp2 = -alpha;
		status1 = hipblasDaxpy(handle1,n,&tmp2,Ap,1,r,1);
		if(status1 != HIPBLAS_STATUS_SUCCESS){
			CLEANUP("compute axpy failed(r)");
			return EXIT_FAILURE;
		}			

		rhop =rho;
		status1 = hipblasDnrm2(handle1,n,r,1,&rho);
		if(status1 != HIPBLAS_STATUS_SUCCESS){
			CLEANUP("compute norm2 failed(rho)");
			return EXIT_FAILURE;
		}
		if(rho<tol){
			break;
		}
		rho = rho*rho;

		//beta = rho/rhop;
		beta = rho/rhop;

		//p = r + beta*p
		status1 = hipblasDscal(handle1,n,&beta,p,1);
		if(status1 != HIPBLAS_STATUS_SUCCESS){
			CLEANUP("compute scal failed(p)");
			return EXIT_FAILURE;
		}	
		status1 = hipblasDaxpy(handle1,n,&done,r,1,p,1);
		if(status1 != HIPBLAS_STATUS_SUCCESS){
			CLEANUP("compute axpy failed(p)");
			return EXIT_FAILURE;
		}			
	}

	//copy x to h_x, and output h_x
	cudaStat1 = hipMemcpy(h_x,x,(size_t)(n*sizeof(double)),hipMemcpyDeviceToHost);
	if(cudaStat1!=hipSuccess)
	{
		CLEANUP("memcpy from device to host failed(x)");
		return EXIT_FAILURE;
	}
	for (int i = 0; i < n; ++i)
	{
		printf("%f\n", h_x[i]);
	}
	CLEANUP("Success");
	return 0;
}
